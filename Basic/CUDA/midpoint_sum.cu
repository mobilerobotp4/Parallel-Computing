#include "hip/hip_runtime.h"
//File name: midpoint_sum.cu
// Numerical Integration Using the Midpoint Rule using Unified memory architecture of Cuda
//input n = no. of term, Block_count = No. of Block, Thread_count = No. of thread
//Output: Summantion of the series
//      : Time taken to execute the program
//Compile: $ nvcc -O2 -arch=sm_30 -o midpoint_sum midpoint_sum.cu
//Run: $srun -p gpu --gres gpu:1 -n 1 -N 1 --pty --mem 1000 -t 3:00 --reservation=cscgpu bash
// $ ./midpoint_sum 16 2 4
// Written By Saswati Bhattacharjee 

//Unified midpoint sum
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include "timer.h"
//Serial sum function prototype declaration
double midpoint(double h_lowerlimit, double h_upperlimit, int h_interval);

__global__ void midpoint_sum(float *a, float *c, float*sum, float step,  int n){


        int tid;
        float local_sum=0.0;
        //Get global thread ID
        tid = blockDim.x*blockIdx.x+threadIdx.x;
        //This program will run if we assign 2 tasks per thread
        //Need to remove the commented portion as per requirement
        int loopstart = 2*tid; //assign 2 task per thread
        int loopend = loopstart+1;
        /* Assign 16 tasks per thread
        int loopstart = 16*tid;
        int loopend = loopstart+15;*/
        /* Assign 256 tasks per thread
           int loopstart=256*tid;
           int loopend = loopstart+255;*/
        /*Assign one task to one thread
          int loopstart = 1*tid;
          int loopend = loopstart+0;*/


        if(tid<n){
        for(int i=loopstart;i<=loopend;i++){
                local_sum+=(step)*(4.0/(1.0+(a[i]*a[i])));

        }

        c[tid]=local_sum;
                atomicAdd(sum,c[tid]);
        }
}

int main(int argc, char* argv[]){

        int thread_count, block_count,n;
        n = strtol(argv[1],NULL,10);
        float *a,  *c, *sum;
        float interval_length[n];
        float mid_values[n-1];
        int j,m; //counter variables
        float lower_limit=0.0, upper_limit=1.0;
        float step;
        double start,finish;

        step =(float) ((upper_limit - lower_limit)/n);


        for( j=0; j<=n;j++){
                interval_length[j] = lower_limit;
                lower_limit= lower_limit + step;

        }

        for(m=0; m<n; m++){
                mid_values[m]=(interval_length[m]+interval_length[m+1])/2.0;
        }
        block_count = strtol(argv[2],NULL,10);
		    thread_count = strtol(argv[3], NULL, 10);
        //shared memory managment
        hipMallocManaged(&a, n*sizeof(float));
        hipMallocManaged(&c, n*sizeof(float));
        hipMallocManaged(&sum, n*sizeof(float));
			for(int i=0; i<n; i++){

              		a[i]= mid_values[i];
			}

		GET_TIME(start);

//Launch Kernel

        midpoint_sum<<<block_count,thread_count>>>(a, c,sum,step, n);

        //Synchronize threads
        hipDeviceSynchronize();
	 GET_TIME(finish);
			
        printf("\nCuda_parallel_sum=%f\n",*sum);
                        hipFree(a);
                        hipFree(c);
                        hipFree(sum);

       printf("\nElapsed time for cuda=%e seconds\n", finish-start);

      double I_value,h_start,h_finish;//variables to execute serial code
      GET_TIME(h_start);
		  I_value = midpoint(0.0,1.0,16); //serial code execution
      GET_TIME(h_finish);
      printf("\nThe answer of the serial_integration is %lf\n", I_value);
      printf("\nElapsed time for host=%e seconds\n", h_finish-h_start);

        return 0;
}
//Serial code function
double midpoint(double h_a, double h_b, int nk){
    double h_interval_length[nk];
    double h_mid[nk-1];
    double h_interval,h_sum, h_partial_sum=0.0;
    h_interval = (h_b-h_a)/(double)nk;

// If lowerlimit=0, upper limit =1 and f[lowerlimit, upperlimit] is divided into  n  subintervals, each of length (upperlimit-lowerlimit)/n. Store the length of the subinterval into interval_length[] array.

                for(int h_i= 0; h_i<=nk; h_i++){
        				h_interval_length[h_i]= h_a;
        				h_a =h_a + h_interval;
    				}
			//Store the value of the midpoints into mid array
    				for(int h_i=0; h_i<nk; h_i++){
        					h_mid[h_i] = (h_interval_length[h_i]+h_interval_length[h_i+1])/2.0;
			// calculate the intermediate sum
        					h_partial_sum = h_partial_sum + (1/(1+ pow(h_mid[h_i],2)));
				}

    			h_sum = (1.0/16.0)*(4.0)*h_partial_sum;
    return h_sum;
}
